#include "hip/hip_runtime.h"
// #######################################################
//
// Exemplo (template) de multiplicação de matrizes em CUDA
// Disciplina: OPRP001 - Programação Paralela
// Prof.: Mauricio Pillon
//
// #######################################################
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>
#include "toolsv3.h"

//isso tem q tirar
#define N 4
#define T 1024 // numero max de threads por bloco

// GPU: Multiplicação das matrizes (a) e (b), resultado em (c)
__global__ void matMult (int *da, int *db, int *dc, int *C_dev) {
    // TODO: Alunos
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    int soma =0;

    for(int ii=0; ii< *C_dev ; ii++){

      printf("[%d]= da: %d db:%d\n",i*blockDim.y+j, i*(*C_dev)+ii, ii*(blockDim.y) + j);
      soma += da[i*(*C_dev)+ii] * db[ii*blockDim.y+j];
    }

    dc[i*blockDim.y+j] = soma;


}

// GPU: Imprime índices na matriz
__global__ void printIndex (void) {
   int i = blockIdx.x * blockDim.x + threadIdx.x;
   int j = blockIdx.y * blockDim.y + threadIdx.y;

   printf ("[%d][%d]=%d\t(x)\t%d\t%d\t%d\t(y)\t%d\t%d\t%d\n",i,j,(i*N+j), threadIdx.x, blockIdx.x, blockDim.x,threadIdx.y, blockIdx.y, blockDim.y);
}


// GPU: Inicializa os vetores (a), (b) e (c) na Memória Global
__global__ void dirtyMem (int *da) {
   int i = blockIdx.x * blockDim.x + threadIdx.x;

   da[i] = 0;
}

// CPU: Inicializa os vetores (a) e (b)
__host__ void initvet(int *host_a, mymatriz mat_a) {
  int lin_a =mat_a.lin, col_a=mat_a.col;

  //iniciar host_a
  for (int i=0; i < lin_a; i++) {
    for (int j=0; j < col_a; j++) {
      printf("host[%d] = %d\n", i*col_a+j, mat_a.matriz[i][j]);
     host_a[i*col_a+j] = mat_a.matriz[i][j];
    }
  }

}

// CPU: Imprime matriz
__host__ void printMat (int *mat, int lin, int col){

	for (int j =0; j < lin; j++)
	printf("\t(%d)", j);
	printf("\n");
	for (int i=0; i < lin; i++) {
		printf("(%d)", i);
		for (int j=0; j < col; j++){
			printf("\t%d", mat[i*col+j]);
		}
		printf("\n");
	}
}


// CPU: função principal
int main(int argc, char const *argv[]) {

  //Lendo matrizes de arquivo, utilizar os arquivos do gerador de arquivos para ter as matrizes
// %%%%%%%%%%%%%%%%%%%%%%%% BEGIN %%%%%%%%%%%%%%%%%%%%%%%%
  // DECLARAÇÃO de VARIÁVEIS
  mymatriz mat_a, mat_b;
  // char filename[100];
  FILE *fmat;
  int nr_line;
  int *vet_line = NULL;
  int L, C, Ca, Lb;
  // double start_time, end_time;
  // %%%%%%%%%%%%%%%%%%%%%%%% END %%%%%%%%%%%%%%%%%%%%%%%%

  if (argc != 3){
		printf ("ERRO: Numero de parametros %s <matriz_a> <matriz_b>\n", argv[0]);
		exit (1);
	}

  fmat = fopen(argv[1],"r");

  if (fmat == NULL) {
      printf("Error: Na abertura dos arquivos.");
      exit(1);
    }

    extrai_parametros_matriz(fmat, &L, &Ca, &vet_line, &nr_line);
    //return 1;
    mat_a.matriz = NULL;
    mat_a.lin = L;
    mat_a.col = Ca;
    if (malocar(&mat_a)) {
      printf ("ERROR: Out of memory\n");
    }
    filein_matriz (mat_a.matriz, L, Ca, fmat, vet_line, nr_line);
    free (vet_line);
    fclose(fmat);
    // %%%%%%%%%%%%%%%%%%%%%%%% END %%%%%%%%%%%%%%%%%%%%%%%%

    // %%%%%%%%%%%%%%%%%%%%%%%% BEGIN %%%%%%%%%%%%%%%%%%%%%%%%

    //               Leitura da Matriz B (arquivo)
    fmat = fopen(argv[2],"r");
    if (fmat == NULL) {
      printf("Error: Na abertura dos arquivos.");
      exit(1);
    }

    extrai_parametros_matriz(fmat, &Lb, &C, &vet_line, &nr_line);
    mat_b.matriz = NULL;
    mat_b.lin = Lb;
    mat_b.col = C;
    if (malocar(&mat_b)) {
      printf ("ERROR: Out of memory\n");
    }
    filein_matriz (mat_b.matriz, Lb, C, fmat, vet_line, nr_line);
    free (vet_line);
    fclose(fmat);
    // %%%%%%%%%%%%%%%%%%%%%%%% END %%%%%%%%%%%%%%%%%%%%%%%% (copiado)

  if(Ca != Lb){
    printf("Error: Matrizes incompativeis\n");
    exit(1);
  }

  int *a, *b, *c; //matrizes (vetores) em host
  int *dev_a, *dev_b, *dev_c;//matrizes (vetores) em device
  int size_a, size_b, size_c;

  printf("R1\n");
  // Alocação de matriz quadrada
  size_a = L * Ca * sizeof(int);
  size_b = Lb * C * sizeof(int);
  size_c = L * C * sizeof(int);
  printf("R2\n");

  printf("R3\n");

  hipError_t error_c;

  // // Alocação de memória no host
  error_c = hipHostMalloc((void **) &a, size_a);
  if(error_c != hipSuccess)
  {
     printf("GPUassert: %s\n", hipGetErrorString(error_c));
  }

  error_c=hipHostMalloc((void **) &b, size_b);
  if(error_c != hipSuccess)
  {
     printf("GPUassert: %s\n", hipGetErrorString(error_c));
  }

  error_c=hipHostMalloc((void **) &c, size_c);
  if(error_c != hipSuccess)
  {
     printf("GPUassert: %s\n", hipGetErrorString(error_c));
  }

  printf("R4\n");

  printf("B1\n");
  initvet(a, mat_a);
  initvet(b, mat_b);
  printf("B2\n");

  // Impressão na tela dos valores dos vetores
  printf ("\t ### Valores Lidos de arquivo na CPU ###\n");
  printf ("\t ### Matriz (a) ### \n");
  printMat(a, L, Ca);
  printf ("\t ### Matriz (b) ### \n");
  printMat(b,Lb, C);

  // Alocação de memória na GPU para os vetores (a,b e c)
  hipMalloc ((void **) &dev_a, size_a);
  hipMalloc ((void **) &dev_b, size_b);
  hipMalloc ((void **) &dev_c, size_c);

  // Atribui valores iniciais aos vetores em GPU
  dirtyMem<<<1, L*Ca>>>(dev_a);
  dirtyMem<<<1, C*Lb>>>(dev_b);
  dirtyMem<<<1, C*L>>>(dev_c);

  ///////////////////////////// teste ///////////////////////////////////////
  //verificar os indices
  // printf("Indice A\n");
  // printIndex<<<1, L*Ca>>>();
  // printf("\nIndice B\n");
  // printIndex<<<1, C*Lb>>>();
  // printf("\nIndice C\n");
  // printIndex<<<1, L*C>>>();
  ////Cópia GPU para CPU
  // hipMemcpy (a, dev_a, size_a, hipMemcpyDeviceToHost);
  // hipMemcpy (b, dev_b, size_b, hipMemcpyDeviceToHost);
  // hipMemcpy (c, dev_c, size_c, hipMemcpyDeviceToHost);
  ////Impressão na tela dos valores dos vetores
  // printf ("\t ### Valores Inicializados na GPU ###\n");
  // printf ("\t ### Matriz (a) ### \n");
  // printMat(a, L, Ca);
  // printf ("\t ### Matriz (b) ### \n");
  // printMat(b,Lb, C);
  // printf ("\t ### Matriz (c) ### \n");
  // printMat(c, L, C);
  ///////////////////////////// teste ///////////////////////////////////////


  //Cópia dos vetores gerados em CPU p/ memória da GPU
  //hipMemcpy(destino, origem, size, direcao)
  hipMemcpy (dev_a, a, size_a, hipMemcpyHostToDevice);
  hipMemcpy (dev_b, b, size_b, hipMemcpyHostToDevice);

  // int L_max = L, C_max = C;
  // if(Lb > L){
  //   L_max = Lb;
  // }
  // if(Ca > C){
  //   C_max = Ca;
  // }

  ////////////////////////////////// Arrumar aqui /////////////////////////////////////////
  //alocar no device o tamanho das matrizes
  // int *linA_dev=hipMalloc((void **) &linA_dev, sizeof(int)), *colA_dev=hipMalloc((void **) &colA_dev, sizeof(int)), *linB_dev=hipMalloc((void **) &linB_dev, sizeof(int)), *colB_dev=hipMalloc((void **) &colB_dev, sizeof(int));

  //Número de blocos e threads p/ dimensões (x,y)
  dim3 dimBlock (1, 1); //dimensao de um bloco (1,1) = 65k x 65k (threads)
  dim3 dimThreads(L, C);//assim podemos multiplicar ate 65k x 65k (pelo q entendi)
  int *C_dev;
  hipMalloc((void **) &C_dev, sizeof(int));
  hipMemcpy (C_dev, &Ca, sizeof(int), hipMemcpyHostToDevice);

  // Imprime as posições acessadas pelo dimBlock e dimThreads
  printIndex<<< dimBlock, dimThreads>>>();

  // // Execução do kernel matMult em GPU
  matMult<<< dimBlock, dimThreads>>>(dev_a, dev_b, dev_c, C_dev);
  hipDeviceSynchronize();

  ///////////////////////////////////////////////////////////////////////////////////////
  // Cópia do vetor (c) da GPU (Memória Global) para CPU
  hipMemcpy (c, dev_c, size_c, hipMemcpyDeviceToHost);

  //Impressão na tela dos valores dos vetores
  printf ("\t ### Valores após processamento em GPU ###\n");
  printf ("\t ### Matriz (a) ### \n");
  printMat(a, L, Ca);
  printf ("\t ### Matriz (b) ### \n");
  printMat(b,Lb, C);
  printf ("\t ### Matriz (c) ### \n");
  printMat(c, L, C);

  // Libera a Memória Global (GPU)
  hipFree(dev_a);
  hipFree(dev_b);
  hipFree(dev_c);
  printf("A1\n");
  // Libera a Memória Global (CPU)
  hipHostFree(a);
  hipHostFree(b);
  hipHostFree(c);
  printf("A2\n");
  return 0;
}
