#include "hip/hip_runtime.h"
// #######################################################
//
// Exemplo (template) de multiplicação de matrizes em CUDA
// Disciplina: OPRP001 - Programação Paralela
// Prof.: Mauricio Pillon
//
// #######################################################
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>
#include "toolsv3.h"

//isso tem q tirar
#define N 4
#define T 2 // numero max de threads por bloco

// GPU: Multiplicação das matrizes (a) e (b), resultado em (c)
__global__ void matMult (int *da, int *db, int *dc, int *C_dev) {
    // TODO: Alunos
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

//int i=threadIdx.x;
//int j=threadIdx.y;

    if(i<blockDim.x && j<blockDim.y){
// printf("i:%d j:%d | bcDim.x:%d  bkDim.y:%d | blockId.x:%d bkId.y:%d\n", i, j,blockDim.x,blockDim.y,blockIdx.x,blockIdx.y );
	int soma=0;
	for(int ii=0; ii< *C_dev ; ii++){

      		printf("[%d]= da: %d db:%d\n",i*blockDim.y+j, i*(*C_dev)+ii, ii*(blockDim.y) + j + blockIdx.y);
	      	soma += da[i*(*C_dev)+ii] * db[ii*blockDim.y+j];
    	}

	dc[i*blockDim.y+j] = soma;
    }


}

// GPU: Imprime índices na matriz
__global__ void printIndex (void) {
   int i = blockIdx.x * blockDim.x + threadIdx.x;
   int j = blockIdx.y * blockDim.y + threadIdx.y;

//	int i= threadIdx.x;
//	int j=threadIdx.y;

   printf ("[%d][%d]=%d\t(x)\t%d\t%d\t%d\t(y)\t%d\t%d\t%d\n",i,j,(i*N+j), threadIdx.x, blockIdx.x, blockDim.x,threadIdx.y, blockIdx.y, blockDim.y);
}


// GPU: Inicializa os vetores (a), (b) e (c) na Memória Global
__global__ void dirtyMem (int *da) {
   int i = blockIdx.x * blockDim.x + threadIdx.x;

   da[i] = 0;
}

// CPU: Inicializa os vetores (a) e (b)
__host__ void initvet(int *host_a, mymatriz mat_a) {
  int lin_a =mat_a.lin, col_a=mat_a.col;

  //iniciar host_a
  for (int i=0; i < lin_a; i++) {
    for (int j=0; j < col_a; j++) {
      // printf("host[%d] = %d\n", i*col_a+j, mat_a.matriz[i][j]);
     host_a[i*col_a+j] = mat_a.matriz[i][j];
    }
  }

}

// CPU: Imprime matriz
__host__ void printMat (int *mat, int lin, int col){

	for (int j =0; j < lin && j<15; j++)
	printf("\t(%d)", j);
	printf("\n");
	for (int i=0; i < lin && i<15; i++) {
		printf("(%d)", i);
		for (int j=0; j < col && j<15; j++){
			printf("\t%d", mat[i*col+j]);
		}
		printf("\n");
	}
}

__host__ mymatriz *mmultiplicar (mymatriz *mat_a, mymatriz *mat_b) {
	//refazer com as 8 permutacoes, para fim de comparacao de desempenho (tempo de execucao)
	//6 permutacoes + 2 tipos com transposta AxBt e AtxB (com dois fors multiplicando linha por linha== ou coluna por coluna)
	mymatriz *mat_c = NULL;

	if (mat_a->col != mat_b->lin){
		printf ("Erro: Matrizes incompatíveis!\n");
		exit(1);
	}

	mat_c = (mymatriz *) malloc (sizeof(mymatriz));

	mat_c->lin = mat_a->lin;
	mat_c->col = mat_b->col;

	if (malocar(mat_c)) {	printf ("ERROR: Out of memory\n"); }

		//a(linhas) x b(colunas)
		printf("Multiplicando com ijk\n" );
		for (int i=0; i <mat_a->lin; i++){
			for (int j=0; j <mat_b->col; j++){
				mat_c->matriz[i][j]=0;
				for (int k=0; k < mat_b->lin; k++){
					mat_c->matriz[i][j] += mat_a->matriz[i][k]*mat_b->matriz[k][j];
				}
			}
		}

    return mat_c;
}

__host__ int mcomparar (mymatriz *mat_a, int *vet_c, int col){
	for (int j =0; j < mat_a->col; j++)
	for (int i=0; i < mat_a->lin; i++) {
		for (int j=0; j < mat_a->col; j++){
			if (mat_a->matriz[i][j] != vet_c[i*col+j]) {
				printf("O elemento [%d,%d] é diferente nas matrizes analisadas!", i,j);
				return 1;
			}
		}
	}
	printf("\tVERIFICADO: Matrizes identicas\n");
	return 0;
}

// %%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%
__host__ int mimprimir (mymatriz *matriz){
	int linha, coluna;
	linha = matriz->lin;
	coluna = matriz->col;

	if (linha > 15) {
		linha = 15;
	}

	if (coluna > 15) {
		coluna = 15;
	}

	for (int j =0; j < coluna; j++)
	printf("\t(%d)", j);
	printf("\n");
	for (int i=0; i < linha; i++) {
		printf("(%d)", i);
		for (int j=0; j < coluna; j++){
			printf("\t%d", matriz->matriz[i][j]);
		}
		printf("\n");
	}

	printf("\n \
	// %%%%%%%%%%%% %%%%%%%%%%%% %%%%%%%%%%%% %%%%%%%%%%%% %%%%%%%%%%%% %%%%%%%%%%%% %%%%%%%%%%%% %%%%%%%%%%%%\n \
	// 	WARNING: Impressão truncada em 15x15! \n \
	// 	WARNING: Último elemento matriz[%d][%d] = %d \n \
	// %%%%%%%%%%%% %%%%%%%%%%%% %%%%%%%%%%%% %%%%%%%%%%%% %%%%%%%%%%%% %%%%%%%%%%%% %%%%%%%%%%%% %%%%%%%%%%%%\n", \
	matriz->lin-1, matriz->col-1, matriz->matriz[matriz->lin-1][matriz->col-1]);
	return 0;
}

// CPU: função principal
int main(int argc, char const *argv[]) {

  //Lendo matrizes de arquivo, utilizar os arquivos do gerador de arquivos para ter as matrizes
// %%%%%%%%%%%%%%%%%%%%%%%% BEGIN %%%%%%%%%%%%%%%%%%%%%%%%
  // DECLARAÇÃO de VARIÁVEIS
  mymatriz mat_a, mat_b;
  // char filename[100];
  FILE *fmat;
  int nr_line;
  int *vet_line = NULL;
  int L, C, Ca, Lb;
  // double start_time, end_time;
  // %%%%%%%%%%%%%%%%%%%%%%%% END %%%%%%%%%%%%%%%%%%%%%%%%

  if (argc != 3){
		printf ("ERRO: Numero de parametros %s <matriz_a> <matriz_b>\n", argv[0]);
		exit (1);
	}

  fmat = fopen(argv[1],"r");

  if (fmat == NULL) {
      printf("Error: Na abertura dos arquivos.");
      exit(1);
    }

    extrai_parametros_matriz(fmat, &L, &Ca, &vet_line, &nr_line);
    //return 1;
    mat_a.matriz = NULL;
    mat_a.lin = L;
    mat_a.col = Ca;
    if (malocar(&mat_a)) {
      printf ("ERROR: Out of memory\n");
    }
    filein_matriz (mat_a.matriz, L, Ca, fmat, vet_line, nr_line);
    free (vet_line);
    fclose(fmat);
    // %%%%%%%%%%%%%%%%%%%%%%%% END %%%%%%%%%%%%%%%%%%%%%%%%

    // %%%%%%%%%%%%%%%%%%%%%%%% BEGIN %%%%%%%%%%%%%%%%%%%%%%%%

    //               Leitura da Matriz B (arquivo)
    fmat = fopen(argv[2],"r");
    if (fmat == NULL) {
      printf("Error: Na abertura dos arquivos.");
      exit(1);
    }

    extrai_parametros_matriz(fmat, &Lb, &C, &vet_line, &nr_line);
    mat_b.matriz = NULL;
    mat_b.lin = Lb;
    mat_b.col = C;
    if (malocar(&mat_b)) {
      printf ("ERROR: Out of memory\n");
    }
    filein_matriz (mat_b.matriz, Lb, C, fmat, vet_line, nr_line);
    free (vet_line);
    fclose(fmat);
    // %%%%%%%%%%%%%%%%%%%%%%%% END %%%%%%%%%%%%%%%%%%%%%%%% (copiado)

  if(Ca != Lb){
    printf("Error: Matrizes incompativeis\n");
    exit(1);
  }

  //multiplicando e tomando o tempo
  printf("Multiplicacao sequencial\n");
  double tempo_s=wtime();
  mymatriz *mult_sequencial = mmultiplicar(&mat_a, &mat_b);
  tempo_s = wtime()-tempo_s;

  int *a, *b, *c; //matrizes (vetores) em host
  int *dev_a, *dev_b, *dev_c;//matrizes (vetores) em device
  int size_a, size_b, size_c;

  // Alocação de matriz quadrada
  size_a = L * Ca * sizeof(int);
  size_b = Lb * C * sizeof(int);
  size_c = L * C * sizeof(int);

  hipError_t error_c;
  // // Alocação de memória no host
  error_c = hipHostMalloc((void **) &a, size_a);
  if(error_c != hipSuccess)
  {
     printf("GPUassert: %s\n", hipGetErrorString(error_c));
  }

  error_c=hipHostMalloc((void **) &b, size_b);
  if(error_c != hipSuccess)
  {
     printf("GPUassert: %s\n", hipGetErrorString(error_c));
  }

  error_c=hipHostMalloc((void **) &c, size_c);
  if(error_c != hipSuccess)
  {
     printf("GPUassert: %s\n", hipGetErrorString(error_c));
  }

  initvet(a, mat_a);
  initvet(b, mat_b);

//  mcomparar(&mat_a, a, Ca);
//  mcomparar(&mat_b, b, C);

  // Impressão na tela dos valores dos vetores
  printf ("\t ### Valores Lidos de arquivo na CPU ###\n");
  printf ("\t ### Matriz (a) ### \n");
  printMat(a, L, Ca);
  printf ("\t ### Matriz (b) ### \n");
  printMat(b,Lb, C);

  // Alocação de memória na GPU para os vetores (a,b e c)
  hipMalloc ((void **) &dev_a, size_a);
  hipMalloc ((void **) &dev_b, size_b);
  hipMalloc ((void **) &dev_c, size_c);

  ////////////////////////////////// Arrumar aqui /////////////////////////////////////////
  //Número de blocos e threads p/ dimensões (x,y)
  dim3 dimBlock (1, 1); //dimensao de um bloco (1,1) = 1024 (threads)
  dim3 dimThreads(L, Ca);//assim podemos multiplicar ate L*C=1024 (pelo q entendi)

  if(L*Ca > T){ // Dimensoes ultrapassam a quantidade de threads de um so bloco
        dimThreads.x=T;
        dimThreads.y=T;

        dimBlock.x= (int) ceil(double(L)/double(T));
        dimBlock.y= (int) ceil(double(Ca)/double(T));
  }

  // Atribui valores iniciais aos vetores em GPU
  dirtyMem<<<dimBlock, dimThreads>>>(dev_a);
////////////////////////////////////////////////////////////
	 dimThreads.x=Lb;
         dimThreads.y=C;
         dimBlock.x= 1;
         dimBlock.y= 1;
   if(Lb*C > T){ // Dimensoes ultrapassam a quantidade de threads de um so bloco
        dimThreads.x=T;
        dimThreads.y=T;

        dimBlock.x= (int) ceil(double(Lb)/double(T));
        dimBlock.y= (int) ceil(double(C)/double(T));
  }

  dirtyMem<<<dimBlock, dimThreads>>>(dev_b);
/////////////////////////////////////////////////////////
	  dimThreads.x=L;
          dimThreads.y=C;
          dimBlock.x= 1;
          dimBlock.y= 1;

   if(L*C > T){ // Dimensoes ultrapassam a quantidade de threads de um so bloco
        dimThreads.x=T;
        dimThreads.y=T;

        dimBlock.x= (int) ceil(double(L)/double(T));
        dimBlock.y= (int) ceil(double(C)/double(T));
  }

  dirtyMem<<<dimBlock, dimThreads>>>(dev_c);

  //Cópia dos vetores gerados em CPU p/ memória da GPU
  //hipMemcpy(destino, origem, size, direcao)
  hipMemcpy (dev_a, a, size_a, hipMemcpyHostToDevice);
  hipMemcpy (dev_b, b, size_b, hipMemcpyHostToDevice);

  ////////////////////////////////// Arrumar aqui /////////////////////////////////////////
  //Número de blocos e threads p/ dimensões (x,y)
    dimThreads.x=L;
    dimThreads.y=C;
    dimBlock.x= 1;
    dimBlock.y= 1;

  if(L*C > T){ // Dimensoes ultrapassam a quantidade de threads de um so bloco
	dimThreads.x=T;
	dimThreads.y=T;

	dimBlock.x= (int) ceil(double(L)/double(T));
	dimBlock.y= (int) ceil(double(C)/double(T));
  }

  int *C_dev;
  hipMalloc((void **) &C_dev, sizeof(int));
  hipMemcpy (C_dev, &Ca, sizeof(int), hipMemcpyHostToDevice);

  // Imprime as posições acessadas pelo dimBlock e dimThreads
  printIndex<<< dimBlock, dimThreads>>>();
   
  ///////////////////////////////////////////////////////////////////////////////////////
  // Execução do kernel matMult em GPU
  printf("Multiplicacao CUDA\n");
  double tempo_c = wtime();
  matMult<<< dimBlock, dimThreads>>>(dev_a, dev_b, dev_c, C_dev);
  hipDeviceSynchronize();
  // Cópia do vetor (c) da GPU (Memória Global) para CPU
  hipMemcpy (c, dev_c, size_c, hipMemcpyDeviceToHost);
  tempo_c = wtime()- tempo_c;
  ///////////////////////////////////////////////////////////////////////////////////////

  printf ("\t ### [CUDA] Matriz (c) ### \n");
  printMat(c, L, C);

  printf("\n\t ### [sequencial] Matriz ###\n");
  mimprimir(mult_sequencial);

  printf("Comparando as matrizes:\n");
  mcomparar(mult_sequencial, c, C);
  printf("Tempo de execucao sequencial: %.3f\n", tempo_s);
  printf("Tempo de execucao CUDA: %.3f\n", tempo_c);

  // Libera a Memória Global (GPU)
  hipFree(dev_a);
  hipFree(dev_b);
  hipFree(dev_c);

  // Libera a Memória Global (CPU)
  hipHostFree(a);
  hipHostFree(b);
  hipHostFree(c);

  return 0;
}
