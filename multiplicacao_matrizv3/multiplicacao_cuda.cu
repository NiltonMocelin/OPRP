#include "hip/hip_runtime.h"
// #######################################################
//
// Exemplo (template) de multiplicação de matrizes em CUDA
// Disciplina: OPRP001 - Programação Paralela
// Prof.: Mauricio Pillon
//
// #######################################################
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>
#include "toolsv3.h"

#define T 1024 // numero max de threads por bloco

// GPU: Multiplicação das matrizes (a) e (b), resultado em (c)
__global__ void matMult (int *da, int *db, int *dc, int *C_dev, int *Cb_dev) {
    // TODO: Alunos
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    if(i<*Cb_dev && j<*Cb_dev){
	int soma=0;

	for(int ii=0; ii< *C_dev ; ii++){

	      	soma += da[i*(*C_dev)+ii] * db[ii*(*Cb_dev)+j];
    	}
	
	dc[i*(*Cb_dev)+j] = soma;
    }


}

// GPU: Imprime índices na matriz
__global__ void printIndex (void) {
   int i = blockIdx.x * blockDim.x + threadIdx.x;
   int j = blockIdx.y * blockDim.y + threadIdx.y;

   printf ("[%d][%d]\t(x)\t%d\t%d\t%d\t(y)\t%d\t%d\t%d\n",i,j, threadIdx.x, blockIdx.x, blockDim.x,threadIdx.y, blockIdx.y, blockDim.y);
}


// GPU: Inicializa os vetores (a), (b) e (c) na Memória Global
__global__ void dirtyMem (int *da) {
   int i = blockIdx.x * blockDim.x + threadIdx.x;

   da[i] = 0;
}

// CPU: Inicializa os vetores (a) e (b)
__host__ void initvet(int *host_a, mymatriz mat_a) {
  int lin_a =mat_a.lin, col_a=mat_a.col;

  //iniciar host_a
  for (int i=0; i < lin_a; i++) {
    for (int j=0; j < col_a; j++) {
     host_a[i*col_a+j] = mat_a.matriz[i][j];
    }
  }

}

// CPU: Imprime matriz
__host__ void printMat (int *mat, int lin, int col){

	for (int j =0; j < lin && j<15; j++)
	printf("\t(%d)", j);
	printf("\n");
	for (int i=0; i < lin && i<15; i++) {
		printf("(%d)", i);
		for (int j=0; j < col && j<15; j++){
			printf("\t%d", mat[i*col+j]);
		}
		printf("\n");
	}
}

__host__ mymatriz *mmultiplicar (mymatriz *mat_a, mymatriz *mat_b) {
	mymatriz *mat_c = NULL;

	if (mat_a->col != mat_b->lin){
		printf ("Erro: Matrizes incompatíveis!\n");
		exit(1);
	}

	mat_c = (mymatriz *) malloc (sizeof(mymatriz));

	mat_c->lin = mat_a->lin;
	mat_c->col = mat_b->col;

	if (malocar(mat_c)) {	printf ("ERROR: Out of memory\n"); }

		//a(linhas) x b(colunas)
		printf("Multiplicando com ijk\n" );
		for (int i=0; i <mat_a->lin; i++){
			for (int j=0; j <mat_b->col; j++){
				mat_c->matriz[i][j]=0;
				for (int k=0; k < mat_b->lin; k++){
					mat_c->matriz[i][j] += mat_a->matriz[i][k]*mat_b->matriz[k][j];
				}
			}
		}

    return mat_c;
}

__host__ int mcomparar (mymatriz *mat_a, int *vet_c, int col){
	for (int j =0; j < mat_a->col; j++)
	for (int i=0; i < mat_a->lin; i++) {
		for (int j=0; j < mat_a->col; j++){
			if (mat_a->matriz[i][j] != vet_c[i*col+j]) {
				printf("O elemento [%d,%d] é diferente nas matrizes analisadas!", i,j);
				return 1;
			}
		}
	}
	printf("\tVERIFICADO: Matrizes identicas\n");
	return 0;
}

// %%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%
__host__ int mimprimir (mymatriz *matriz){
	int linha, coluna;
	linha = matriz->lin;
	coluna = matriz->col;

	if (linha > 15) {
		linha = 15;
	}

	if (coluna > 15) {
		coluna = 15;
	}

	for (int j =0; j < coluna; j++)
	printf("\t(%d)", j);
	printf("\n");
	for (int i=0; i < linha; i++) {
		printf("(%d)", i);
		for (int j=0; j < coluna; j++){
			printf("\t%d", matriz->matriz[i][j]);
		}
		printf("\n");
	}

	printf("\n \
	// %%%%%%%%%%%% %%%%%%%%%%%% %%%%%%%%%%%% %%%%%%%%%%%% %%%%%%%%%%%% %%%%%%%%%%%% %%%%%%%%%%%% %%%%%%%%%%%%\n \
	// 	WARNING: Impressão truncada em 15x15! \n \
	// 	WARNING: Último elemento matriz[%d][%d] = %d \n \
	// %%%%%%%%%%%% %%%%%%%%%%%% %%%%%%%%%%%% %%%%%%%%%%%% %%%%%%%%%%%% %%%%%%%%%%%% %%%%%%%%%%%% %%%%%%%%%%%%\n", \
	matriz->lin-1, matriz->col-1, matriz->matriz[matriz->lin-1][matriz->col-1]);
	return 0;
}

// CPU: função principal
int main(int argc, char const *argv[]) {

  //Lendo matrizes de arquivo, utilizar os arquivos do gerador de arquivos para ter as matrizes
// %%%%%%%%%%%%%%%%%%%%%%%% BEGIN %%%%%%%%%%%%%%%%%%%%%%%%
  // DECLARAÇÃO de VARIÁVEIS
  mymatriz mat_a, mat_b;
  // char filename[100];
  FILE *fmat;
  int nr_line;
  int *vet_line = NULL;
  int L, C, Ca, Lb;
  // double start_time, end_time;
  // %%%%%%%%%%%%%%%%%%%%%%%% END %%%%%%%%%%%%%%%%%%%%%%%%

  if (argc != 3){
		printf ("ERRO: Numero de parametros %s <matriz_a> <matriz_b>\n", argv[0]);
		exit (1);
	}

  fmat = fopen(argv[1],"r");

  if (fmat == NULL) {
      printf("Error: Na abertura dos arquivos.");
      exit(1);
    }

    extrai_parametros_matriz(fmat, &L, &Ca, &vet_line, &nr_line);
    //return 1;
    mat_a.matriz = NULL;
    mat_a.lin = L;
    mat_a.col = Ca;
    if (malocar(&mat_a)) {
      printf ("ERROR: Out of memory\n");
    }
    filein_matriz (mat_a.matriz, L, Ca, fmat, vet_line, nr_line);
    free (vet_line);
    fclose(fmat);
    // %%%%%%%%%%%%%%%%%%%%%%%% END %%%%%%%%%%%%%%%%%%%%%%%%

    // %%%%%%%%%%%%%%%%%%%%%%%% BEGIN %%%%%%%%%%%%%%%%%%%%%%%%

    //               Leitura da Matriz B (arquivo)
    fmat = fopen(argv[2],"r");
    if (fmat == NULL) {
      printf("Error: Na abertura dos arquivos.");
      exit(1);
    }

    extrai_parametros_matriz(fmat, &Lb, &C, &vet_line, &nr_line);
    mat_b.matriz = NULL;
    mat_b.lin = Lb;
    mat_b.col = C;
    if (malocar(&mat_b)) {
      printf ("ERROR: Out of memory\n");
    }
    filein_matriz (mat_b.matriz, Lb, C, fmat, vet_line, nr_line);
    free (vet_line);
    fclose(fmat);
    // %%%%%%%%%%%%%%%%%%%%%%%% END %%%%%%%%%%%%%%%%%%%%%%%% (copiado)

  if(Ca != Lb){
    printf("Error: Matrizes incompativeis\n");
    exit(1);
  }

  //multiplicando e tomando o tempo
  printf("Multiplicacao sequencial\n");
  double tempo_s=wtime();
  mymatriz *mult_sequencial = mmultiplicar(&mat_a, &mat_b);
  tempo_s = wtime()-tempo_s;

  int *a, *b, *c; //matrizes (vetores) em host
  int *dev_a, *dev_b, *dev_c;//matrizes (vetores) em device
  int size_a, size_b, size_c;

  // Alocação de matriz quadrada
  size_a = L * Ca * sizeof(int);
  size_b = Lb * C * sizeof(int);
  size_c = L * C * sizeof(int);

  hipError_t error_c;
  // Alocação de memória no host
  error_c = hipHostMalloc((void **) &a, size_a);
  if(error_c != hipSuccess)
  {
     printf("GPUassert: %s\n", hipGetErrorString(error_c));
  }

  error_c=hipHostMalloc((void **) &b, size_b);
  if(error_c != hipSuccess)
  {
     printf("GPUassert: %s\n", hipGetErrorString(error_c));
  }

  error_c=hipHostMalloc((void **) &c, size_c);
  if(error_c != hipSuccess)
  {
     printf("GPUassert: %s\n", hipGetErrorString(error_c));
  }

  initvet(a, mat_a);
  initvet(b, mat_b);

  // Impressão na tela dos valores dos vetores
  printf ("\t ### Valores Lidos de arquivo na CPU ###\n");
  printf ("\t ### Matriz (a) ### \n");
  printMat(a, L, Ca);
  printf ("\t ### Matriz (b) ### \n");
  printMat(b,Lb, C);

  // Alocação de memória na GPU para os vetores (a,b e c)
  hipMalloc ((void **) &dev_a, size_a);
  hipMalloc ((void **) &dev_b, size_b);
  hipMalloc ((void **) &dev_c, size_c);

  ////////////////////////////////// Parte inutil (carregar valores nas posicoes antes de enviar os valores corretos///////////////////////////////////////
  //Número de blocos e threads p/ dimensões (x,y)
  dim3 dimBlock (1, 1); //dimensao de um bloco (1,1) = 1024 (threads)
  dim3 dimThreads(L, Ca);//assim podemos multiplicar ate L*C=1024 (pelo q entendi)

  if(L*Ca > T){ // Dimensoes ultrapassam a quantidade de threads de um so bloco
	dimThreads.x=(int) ceil(sqrt(T));
        dimThreads.y=(int) ceil(sqrt(T));

        dimBlock.x= (int) ceil(double(L)/sqrt(T));//sqrt(1024)
        dimBlock.y= (int) ceil(double(Ca)/sqrt(T));//sqrt(1024) pois 32*32=1024 e precisamos L*C < 1024
  }

  // Atribui valores iniciais aos vetores em GPU
  dirtyMem<<<dimBlock, dimThreads>>>(dev_a);
////////////////////////////////////////////////////////////
	 dimThreads.x=Lb;
         dimThreads.y=C;
         dimBlock.x= 1;
         dimBlock.y= 1;
   if(Lb*C > T){ // Dimensoes ultrapassam a quantidade de threads de um so bloco
	dimThreads.x=(int) ceil(sqrt(T));
        dimThreads.y=(int) ceil(sqrt(T));

        dimBlock.x= (int) ceil(double(Lb)/sqrt(T));//sqrt(1024)
        dimBlock.y= (int) ceil(double(C)/sqrt(T));//sqrt(1024) pois 32*32=1024 e precisamos L*C < 1024
 
 }

  dirtyMem<<<dimBlock, dimThreads>>>(dev_b);
/////////////////////////////////////////////////////////
	  dimThreads.x=L;
          dimThreads.y=C;
          dimBlock.x= 1;
          dimBlock.y= 1;
//////parte inutil menos inutil, mas ainda assim serve pra nada, carregar 0 na matriz resultante//
   if(L*C > T){ // Dimensoes ultrapassam a quantidade de threads de um so bloco
  
	dimThreads.x=(int) ceil(sqrt(T));
        dimThreads.y=(int) ceil(sqrt(T));

        dimBlock.x= (int) ceil(double(L)/sqrt(T));//sqrt(1024)
        dimBlock.y= (int) ceil(double(C)/sqrt(T));//sqrt(1024) pois 32*32=1024 e precisamos L*C < 1024
 }
  dirtyMem<<<dimBlock, dimThreads>>>(dev_c);
////////////////////////////////Fim parte inutil/////////////////////////////////////////////

  //Cópia dos vetores gerados em CPU p/ memória da GPU
  //hipMemcpy(destino, origem, size, direcao)
  hipMemcpy (dev_a, a, size_a, hipMemcpyHostToDevice);
  hipMemcpy (dev_b, b, size_b, hipMemcpyHostToDevice);

  ////////////////////////////////// Arrumar aqui /////////////////////////////////////////
  //Número de blocos e threads p/ dimensões (x,y)
    dimThreads.x=L;
    dimThreads.y=C;
    dimBlock.x= 1;
    dimBlock.y= 1;

  if(L*C > T){ // Dimensoes ultrapassam a quantidade de threads de um so bloco
	dimThreads.x=(int) ceil(sqrt(T));
	dimThreads.y=(int) ceil(sqrt(T));

	dimBlock.x= (int) ceil(double(L)/sqrt(T));//sqrt(1024)
	dimBlock.y= (int) ceil(double(C)/sqrt(T));//sqrt(1024) pois 32*32=1024 e precisamos L*C < 1024
  }

  int *C_dev, *Cb_dev;//c_dev = Ca (matrizA) e Cb_dev=C (matrizb)
  hipMalloc((void **) &C_dev, sizeof(int));
  hipMalloc((void **) &Cb_dev, sizeof(int));
  hipMemcpy (C_dev, &Ca, sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(Cb_dev, &C, sizeof(int), hipMemcpyHostToDevice);

  ///////////////////////////////////////////////////////////////////////////////////////
  // Execução do kernel matMult em GPU
  printf("Multiplicacao CUDA\n");
  double tempo_c = wtime();
  matMult<<< dimBlock, dimThreads>>>(dev_a, dev_b, dev_c, C_dev, Cb_dev);

  hipDeviceSynchronize();

  // Cópia do vetor (c) da GPU (Memória Global) para CPU
  hipMemcpy (c, dev_c, size_c, hipMemcpyDeviceToHost);
  tempo_c = wtime()- tempo_c;
  ///////////////////////////////////////////////////////////////////////////////////////

  printf ("\t ### [CUDA] Matriz (c) ### \n");
  printMat(c, L, C);

  printf("\n\t ### [sequencial] Matriz ###\n");
  mimprimir(mult_sequencial);

  printf("Comparando as matrizes:\n");
  mcomparar(mult_sequencial, c, C);
  printf("Tempo de execucao sequencial: %.3f\n", tempo_s);
  printf("Tempo de execucao CUDA: %.3f\n", tempo_c);

  // Libera a Memória Global (GPU)
  hipFree(dev_a);
  hipFree(dev_b);
  hipFree(dev_c);

  // Libera a Memória Global (CPU)
  hipHostFree(a);
  hipHostFree(b);
  hipHostFree(c);

  return 0;
}
